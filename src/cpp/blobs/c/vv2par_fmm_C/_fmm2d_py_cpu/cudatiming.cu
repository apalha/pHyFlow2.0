#include <stdlib.h>
#include "hip/hip_runtime.h" //why does the code compile without this?
#include "mex.h"
#ifdef _MSC_VER
#define snprintf _snprintf
#endif
//Like cudasafe, but takes two messages
void cudatimingsafe( hipError_t error,const char* basemessage,const char* message)
{
    char errormessage[1000];
    if(error!=hipSuccess) {
        snprintf(errormessage,999,"%s %s : %s (%i) \n",basemessage,message,hipGetErrorString(error),error);
        hipGetLastError(); //remove the error message
        mexErrMsgTxt(errormessage); 
    }
}

//create events
void cudaSafeEventCreate(hipEvent_t *event, const char* str) //only a wrapper to be able to call it from cpp files
{
  cudatimingsafe(hipEventCreate(event),"hipEventCreate", str);
}
//start and stop timer
void cudaSafeEventRecord(hipEvent_t event, const char* str) //only a wrapper to be able to call it from cpp files
{
  cudatimingsafe(hipEventRecord(event),"hipEventRecord", str);
}
//synchronizes with GPU
void cudaSafeEventSynchronize(hipEvent_t event, const char* str) //only a wrapper to be able to call it from cpp files
{
  cudatimingsafe(hipEventSynchronize(event),"hipEventSynchronize", str);
}
//get elapsed time
void cudaSafeEventElapsedTime(float* elapsedtime, hipEvent_t startevent, hipEvent_t stopevent, const char* str) //only a wrapper to be able to call it from cpp files
{
  cudatimingsafe(hipEventElapsedTime(elapsedtime, startevent, startevent),"hipEventElapsedTime", str);
}
//remove event
void cudaSafeEventDestroy(hipEvent_t event, const char* str) //only a wrapper to be able to call it from cpp files
{
  cudatimingsafe(hipEventDestroy(event),"hipEventDestroy", str);
}
//create events and start the first timer
void cudaTimingCreateAndStart(hipEvent_t *startevent, hipEvent_t *stopevent, const char* startstr, const char *stopstr)
{
  cudatimingsafe(hipEventCreate(startevent),"hipEventCreate", startstr);
  cudatimingsafe(hipEventCreate(stopevent),"hipEventCreate", stopstr);
  cudatimingsafe(hipEventRecord(*startevent),"hipEventRecord", startstr);
}
//synchronizes with GPU, and prints the time between start and stop event and removes the two events
void cudaTimingSyncPrintAndDestroy(hipEvent_t startevent, hipEvent_t stopevent,double* timing,int index,int printtime,const char* outputstr, const char* startstr, const char *stopstr)
{
  float elapsedtime;
  cudatimingsafe(hipEventSynchronize(stopevent), "hipEventSynchronize", stopstr);
  cudatimingsafe(hipEventElapsedTime(&elapsedtime, startevent, stopevent), "hipEventElapsedTime",startstr);
  cudatimingsafe(hipEventDestroy(stopevent), "hipEventDestroy",stopstr);
  cudatimingsafe(hipEventDestroy(startevent), "hipEventDestroy",startstr);
  if(printtime)
    mexPrintf("%s: %f\n",outputstr, elapsedtime/1000);
  if(timing!=NULL)
    timing[index]=elapsedtime/1000;
}