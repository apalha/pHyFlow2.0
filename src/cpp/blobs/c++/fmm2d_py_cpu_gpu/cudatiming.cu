#include <stdlib.h>
#include<stdio.h>
#include "hip/hip_runtime.h" //why does the code compile without this?
#ifndef C_CODE          /*if in C file mode, redefine all mex functions to c functions*/
#include "mex.h"
#endif
#ifdef _MSC_VER
#define snprintf _snprintf
#endif

#ifdef C_CODE          /*if in C file mode, redefine all mex functions to c functions*/
#define mxFree free
#define mxMalloc malloc
#define mxCalloc calloc
#define mexPrintf printf
#define mxLogical int
#endif

//Like cudasafe, but takes two messages
void cudatimingsafe( hipError_t error,const char* basemessage,const char* message)
{
    char errormessage[1000];
    if(error!=hipSuccess) {
        snprintf(errormessage,999,"%s %s : %s (%i) \n",basemessage,message,hipGetErrorString(error),error);
        hipGetLastError(); //remove the error message
        //mexErrMsgTxt(errormessage);
        printf("Error message goes here!");
    }
}

//create events
void cudaSafeEventCreate(hipEvent_t *event, const char* str) //only a wrapper to be able to call it from cpp files
{
  cudatimingsafe(hipEventCreate(event),"hipEventCreate", str);
}
//start and stop timer
void cudaSafeEventRecord(hipEvent_t event, const char* str) //only a wrapper to be able to call it from cpp files
{
  cudatimingsafe(hipEventRecord(event),"hipEventRecord", str);
}
//synchronizes with GPU
void cudaSafeEventSynchronize(hipEvent_t event, const char* str) //only a wrapper to be able to call it from cpp files
{
  cudatimingsafe(hipEventSynchronize(event),"hipEventSynchronize", str);
}
//get elapsed time
void cudaSafeEventElapsedTime(float* elapsedtime, hipEvent_t startevent, hipEvent_t stopevent, const char* str) //only a wrapper to be able to call it from cpp files
{
  cudatimingsafe(hipEventElapsedTime(elapsedtime, startevent, startevent),"hipEventElapsedTime", str);
}
//remove event
void cudaSafeEventDestroy(hipEvent_t event, const char* str) //only a wrapper to be able to call it from cpp files
{
  cudatimingsafe(hipEventDestroy(event),"hipEventDestroy", str);
}
//create events and start the first timer
void cudaTimingCreateAndStart(hipEvent_t *startevent, hipEvent_t *stopevent, const char* startstr, const char *stopstr)
{
  cudatimingsafe(hipEventCreate(startevent),"hipEventCreate", startstr);
  cudatimingsafe(hipEventCreate(stopevent),"hipEventCreate", stopstr);
  cudatimingsafe(hipEventRecord(*startevent),"hipEventRecord", startstr);
}
//synchronizes with GPU, and prints the time between start and stop event and removes the two events
void cudaTimingSyncPrintAndDestroy(hipEvent_t startevent, hipEvent_t stopevent,double* timing,int index,int printtime,const char* outputstr, const char* startstr, const char *stopstr)
{
  float elapsedtime;
  cudatimingsafe(hipEventSynchronize(stopevent), "hipEventSynchronize", stopstr);
  cudatimingsafe(hipEventElapsedTime(&elapsedtime, startevent, stopevent), "hipEventElapsedTime",startstr);
  cudatimingsafe(hipEventDestroy(stopevent), "hipEventDestroy",stopstr);
  cudatimingsafe(hipEventDestroy(startevent), "hipEventDestroy",startstr);
  if(printtime)
    mexPrintf("%s: %f\n",outputstr, elapsedtime/1000);
  if(timing!=NULL)
    timing[index]=elapsedtime/1000;
}
